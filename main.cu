#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define size 1000

void matex_CPU(float* A,float* B,float* R)
{
    float h,s;
    for(int i=0;i<size;i++)
    {
        for(int q=0;q<size;q++)
        {
            h = 0;
            for(int k=0;k<size;k++)
            {
                s = A[i*size+k]*B[k*size+q];
                h+=s;
            }
            R[i*size+q] = h;
        }
    }
}

__global__ void matex_GPU(float* A,float* B,float* R)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y; // 行号
    int col = blockIdx.x * blockDim.x + threadIdx.x; // 列号
    
    if(row < size && col < size) { // 确保不超出矩阵边界
        float value = 0;
        for(int k = 0; k < size; ++k) {
            value += A[row * size + k] * B[k * size + col];
        }
        R[row * size + col] = value;
    }
}

int main(void)
{
    srand((unsigned)time(NULL));
    float *A = (float*)malloc(size * size * sizeof(float));
    float *B = (float*)malloc(size * size * sizeof(float));
    float *RC = (float*)malloc(size * size * sizeof(float));
    float *RG = (float*)malloc(size * size * sizeof(float));
    for(int i=0;i<size;i++)
    {
        for(int j=0;j<size;j++)
        {
            A[i*size+j] = float(rand())/100000;
        }
    }
    for(int i=0;i<size;i++)
    {
        for(int j=0;j<size;j++)
        {
            B[i*size+j] = float(rand())/100000;
        }
    }
    float *RGDevice, *ADevice, *BDevice;
    hipMalloc((float**)&RGDevice,sizeof(float)*size*size);
    hipMalloc((float**)&ADevice,sizeof(float)*size*size);
    hipMalloc((float**)&BDevice,sizeof(float)*size*size);
    hipMemcpy(RGDevice,RG,sizeof(float)*size*size,hipMemcpyHostToDevice);
    hipMemcpy(ADevice,A,sizeof(float)*size*size,hipMemcpyHostToDevice);
    hipMemcpy(BDevice,B,sizeof(float)*size*size,hipMemcpyHostToDevice);
    dim3 threadsPerBlock(32, 32); // 每个block中的线程数,最大为1024
    dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (size + threadsPerBlock.y - 1) / threadsPerBlock.y); // block的数量
    matex_GPU<<<numBlocks, threadsPerBlock>>>(ADevice,BDevice,RGDevice);
    matex_CPU(A,B,RC);
    hipDeviceSynchronize();
    hipMemcpy(RG, RGDevice, sizeof(float)*size*size, hipMemcpyDeviceToHost);
    int count = 0;
    for(int i=0;i<size;i++)
    {
        for(int j=0;j<size;j++)
        {
            if ((fabs(RG[i*size+j] - RC[i*size+j]))/ RG[i*size+j] > 0.01) 
            {
                count++;
                break;
            }
        }
        if(count!=0)
        {
            break;
        }
    }
    if(count!=0)
    {
        printf("wrong\n");
    }
    else
    {
        printf("right\n");
    }

    // for(int i=0;i<size;i++)
    // {
    //     for(int j=0;j<size;j++)
    //     {
    //         printf("%lf ",RC[i*size+j]);
    //     }
    //     printf("\n");
    // }
    // printf("\n");
    // for(int i=0;i<size;i++)
    // {
    //     for(int j=0;j<size;j++)
    //     {
    //         printf("%lf ",RG[i*size+j]);
    //     }
    //     printf("\n");
    // }
    free(A);
    free(B);
    free(RC);
    free(RG);
    hipFree(RGDevice);
    hipFree(ADevice);
    hipFree(BDevice);
    return 0;
}
